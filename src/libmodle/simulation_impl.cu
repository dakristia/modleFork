#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <fmt/format.h>

#include <algorithm>
#include <cstdint>
#include <cstdio>
#include <cuda/std/atomic>
#include <vector>

#include "modle/contacts.cuh"
#include "modle/extrusion_barriers.cuh"
#include "modle/simulation.cuh"

namespace modle::cu::Simulation {

__host__ __device__ State::~State() {
  delete[] barrier_pos;
  delete[] barrier_directions;
  delete[] barrier_mask;

  delete[] rev_unit_pos;
  delete[] fwd_unit_pos;
  delete[] lef_rev_unit_idx;
  delete[] lef_fwd_unit_idx;

  delete[] rev_moves_buff;
  delete[] fwd_moves_buff;
  delete[] rev_collision_mask;
  delete[] fwd_collision_mask;

  delete[] lef_unloader_affinities;
  delete[] contact_local_buff;
  delete[] epoch_buff;

  delete[] rng_state;
}

__global__ void init_global_buffers_kernel(State* states, uint32_t nstates, uint32_t max_nlefs,
                                           uint32_t max_nbarriers,
                                           cuda::std::atomic<bool>* global_ok) {
  const auto id = threadIdx.x + blockIdx.x * blockDim.x;
  const auto nthreads = blockDim.x * gridDim.x;

  const auto chunk_size = max(1, (nstates + 1) / nthreads);

  if (id == 0) {
    bool local_ok = true;
    local_ok &= !!(states[0].barrier_pos = new bp_t[max_nbarriers]);
    local_ok &= !!(states[0].barrier_directions = new dna::Direction[max_nbarriers]);
    local_ok &= !!(states[0].barrier_mask = new bool[max_nbarriers]);
    if (!local_ok) {
      *global_ok = false;
    }
  }
  __syncthreads();

  const auto i0 = id * chunk_size;
  const auto i1 = (id + 1) * chunk_size;
  for (auto i = i0; i < i1 && i < nstates && *global_ok; ++i) {
    bool local_ok = true;
    states[i].barrier_pos = states[0].barrier_pos;
    states[i].barrier_directions = states[0].barrier_directions;
    states[i].barrier_mask = states[0].barrier_mask;

    local_ok &= !!(states[i].rev_unit_pos = new bp_t[max_nlefs]);
    local_ok &= !!(states[i].fwd_unit_pos = new bp_t[max_nlefs]);
    local_ok &= !!(states[i].lef_rev_unit_idx = new bp_t[max_nlefs]);
    local_ok &= !!(states[i].lef_fwd_unit_idx = new bp_t[max_nlefs]);

    local_ok &= !!(states[i].rev_moves_buff = new bp_t[max_nlefs]);
    local_ok &= !!(states[i].fwd_moves_buff = new bp_t[max_nlefs]);
    local_ok &= !!(states[i].rev_collision_mask = new bp_t[max_nlefs]);
    local_ok &= !!(states[i].fwd_collision_mask = new bp_t[max_nlefs]);

    local_ok &= !!(states[i].lef_unloader_affinities = new float[max_nlefs]);
    local_ok &= !!(states[i].contact_local_buff = new uint2[max_nlefs]);  // TODO CHANGEME
    local_ok &= !!(states[i].epoch_buff = new bp_t[max_nlefs]);

    local_ok &= !!(states[i].rng_state = new hiprandStatePhilox4_32_10_t[blockDim.x]);

    if (!local_ok) {
      *global_ok = false;
      return;
    }
  }
}

__global__ void free_global_buffers_kernel(State* states, uint32_t nstates) {
  const auto id = threadIdx.x + blockIdx.x * blockDim.x;
  const auto nthreads = blockDim.x * gridDim.x;

  const auto chunk_size = max(1, (nstates + 1) / nthreads);

  if (id == 0) {
    delete[] states[0].barrier_pos;
    delete[] states[0].barrier_directions;
    delete[] states[0].barrier_mask;
  }
  __syncthreads();

  const auto i0 = id * chunk_size;
  const auto i1 = (id + 1) * chunk_size;
  for (auto i = i0; i < i1 && i < nstates; ++i) {
    delete[] states[i].rev_unit_pos;
    delete[] states[i].fwd_unit_pos;
    delete[] states[i].lef_rev_unit_idx;
    delete[] states[i].lef_fwd_unit_idx;

    delete[] states[i].rev_moves_buff;
    delete[] states[i].fwd_moves_buff;
    delete[] states[i].rev_collision_mask;
    delete[] states[i].fwd_collision_mask;

    delete[] states[i].lef_unloader_affinities;
    delete[] states[i].contact_local_buff;
    delete[] states[i].epoch_buff;

    delete[] states[i].rng_state;
  }

  __syncthreads();
  delete[] states;
}

__global__ void setup_curand_kernel(hiprandStatePhilox4_32_10_t* state, uint64_t seed = 123456789) {
  const auto id = threadIdx.x + blockIdx.x * blockDim.x;
  hiprand_init(seed, id, 0, &state[id]);
}

__global__ void init_barriers(ExtrusionBarrier* barriers, size_t nbarriers, const size_t* positions,
                              const dna::Direction* directions) {
  const auto id = threadIdx.x + blockIdx.x * blockDim.x;
  const auto chunk_size = nbarriers / blockDim.x;

  const auto i0 = id * chunk_size;
  const auto i1 = i0 + chunk_size;

  for (auto i = i0; i < i1 && i < nbarriers; ++i) {
    barriers[i] = ExtrusionBarrier{positions[i], 0.93, 0.7, directions[i]};
  }
  __syncthreads();
}

__global__ void init_ctcf_states_kernel(CTCF::State* mask, size_t nctcfs,
                                        hiprandStatePhilox4_32_10_t* rng_state,
                                        double pblock = 0.85) {
  const auto id = threadIdx.x + blockIdx.x * blockDim.x;
  const auto chunk_size = (nctcfs + 1) / blockDim.x;

  auto local_state = rng_state[id];

  const auto i0 = id * chunk_size;
  const auto i1 = i0 + chunk_size;
  auto i = i0;

  do {
    const auto buff = hiprand_uniform2_double(&local_state);
    mask[i++] = buff.x < pblock ? CTCF::OCCUPIED : CTCF::NOT_OCCUPIED;
    if (i < i1) {
      mask[i++] = buff.y < pblock ? CTCF::OCCUPIED : CTCF::NOT_OCCUPIED;
    }
  } while (i < i1 && i < nctcfs);

  rng_state[id] = local_state;
  __syncthreads();
}

__global__ void mykernel() {
  const auto id = threadIdx.x + blockIdx.x * blockDim.x;
  printf("id=%d\n", id);
}

__global__ void mykernel2(ContactMatrix<uint32_t>* m, cuda::std::atomic<uint32_t>* buff,
                          size_t nrows, size_t ncols) {
  const auto id = threadIdx.x + blockIdx.x * blockDim.x;
  if (id == 0) {
    m->reset(buff, nrows, ncols);
  }
  __syncthreads();
  m->add(id, id, static_cast<uint32_t>(id));
}

__global__ void mykernel3(const ExtrusionBarrier* barriers, size_t nbarriers, CTCF::State* mask,
                          hiprandStatePhilox4_32_10_t* rng_state) {
  const auto id = threadIdx.x + blockIdx.x * blockDim.x;
  for (auto i = 0UL; i < 20000; ++i) {
    if (id == 0 && i % 1000 == 0) printf("iter=%lu\n", i);
    CTCF::update_states(barriers, nbarriers, mask, rng_state);
  }
}

std::vector<uint32_t> run_mykernel2(size_t nrows, size_t ncols, size_t& missed_updates,
                                    size_t& tot_contacts) {
  cuda::std::atomic<uint32_t>* dev_buff{nullptr};
  modle::cu::ContactMatrix<uint32_t>* matrix{nullptr};

  if (const auto status = hipMalloc(&matrix, sizeof(ContactMatrix<uint32_t>));
      status != hipSuccess) {
    throw std::runtime_error(
        fmt::format(FMT_STRING("Unable to allocate {} bytes of memory on the device"),
                    sizeof(ContactMatrix<uint32_t>)));
  }
  if (const auto status =
          hipMalloc(&dev_buff, nrows * ncols * sizeof(cuda::std::atomic<uint32_t>));
      status != hipSuccess) {
    throw std::runtime_error(
        fmt::format(FMT_STRING("Unable to allocate {:.2f} MB of memory on the device"),
                    nrows * ncols * sizeof(cuda::std::atomic<uint32_t>)));
  }

  mykernel2<<<1, 10>>>(matrix, dev_buff, nrows, ncols);
  if (const auto status = hipDeviceSynchronize(); status != hipSuccess) {
    throw std::runtime_error("cudaDeviceSyncronize failed");
  }
  missed_updates = 0;
  tot_contacts = 0;

  if (const auto status = hipFree(matrix); status != hipSuccess) {
    throw std::runtime_error("hipFree failed.");
  }

  std::vector<uint32_t> host_buff(nrows * ncols);
  if (const auto status =
          hipMemcpy(host_buff.data(), dev_buff,
                     nrows * ncols * sizeof(cuda::std::atomic<uint32_t>), hipMemcpyDeviceToHost);
      status != hipSuccess) {
    throw std::runtime_error("hipMemcpy failed.");
  }
  if (const auto status = hipFree(dev_buff); status != hipSuccess) {
    throw std::runtime_error("hipFree failed.");
  }

  return host_buff;
}

std::vector<CTCF::State> run_mykernel3(const std::vector<ExtrusionBarrier>& host_barriers,
                                       uint64_t seed) {
  ExtrusionBarrier* dev_barriers{nullptr};
  CTCF::State* dev_barrier_states{nullptr};
  hiprandStatePhilox4_32_10_t* dev_rng_states{nullptr};
  std::vector<size_t> host_pos_buff(host_barriers.size());
  std::vector<dna::Direction> host_dir_buff(host_barriers.size());
  size_t* dev_pos_buff{nullptr};
  dna::Direction* dev_dir_buff{nullptr};

  const auto grid_size = 1UL;
  const auto block_size = 64UL;

  CUDA_CALL(hipMalloc(&dev_barriers, sizeof(ExtrusionBarrier) * host_barriers.size()));
  CUDA_CALL(hipMalloc(&dev_barrier_states, sizeof(CTCF::State) * host_barriers.size()));
  CUDA_CALL(
      hipMalloc(&dev_rng_states, grid_size * block_size * sizeof(hiprandStatePhilox4_32_10_t)));
  CUDA_CALL(hipMalloc(&dev_pos_buff, sizeof(size_t) * host_barriers.size()));
  CUDA_CALL(hipMalloc(&dev_dir_buff, sizeof(CTCF::State) * host_barriers.size()));
  CUDA_CALL(hipDeviceSynchronize());

  std::transform(host_barriers.begin(), host_barriers.end(), host_pos_buff.begin(),
                 [](const auto& b) { return b.pos(); });

  std::transform(host_barriers.begin(), host_barriers.end(), host_dir_buff.begin(),
                 [](const auto& b) {
                   return CTCF::major_blocking_dir_to_motif_dir(b.blocking_direction_major());
                 });

  CUDA_CALL(hipMemcpy(dev_pos_buff, host_pos_buff.data(), sizeof(size_t) * host_barriers.size(),
                       hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(dev_dir_buff, host_dir_buff.data(),
                       sizeof(dna::Direction) * host_barriers.size(), hipMemcpyHostToDevice));

  CUDA_CALL(hipDeviceSynchronize());
  setup_curand_kernel<<<grid_size, block_size>>>(dev_rng_states, seed);
  init_barriers<<<grid_size, block_size>>>(dev_barriers, host_barriers.size(), dev_pos_buff,
                                           dev_dir_buff);
  CUDA_CALL(hipDeviceSynchronize());
  init_ctcf_states_kernel<<<grid_size, block_size>>>(dev_barrier_states, host_barriers.size(),
                                                     dev_rng_states);
  CUDA_CALL(hipDeviceSynchronize());

  mykernel3<<<grid_size, block_size>>>(dev_barriers, host_barriers.size(), dev_barrier_states,
                                       dev_rng_states);
  CUDA_CALL(hipDeviceSynchronize());
  CUDA_CALL(hipFree(dev_barriers));
  CUDA_CALL(hipFree(dev_rng_states));

  std::vector<CTCF::State> host_states(host_barriers.size());
  CUDA_CALL(hipMemcpy(host_states.data(), dev_barrier_states, host_states.size(),
                       hipMemcpyDeviceToHost));
  CUDA_CALL(hipDeviceSynchronize());
  CUDA_CALL(hipFree(dev_barrier_states));
  CUDA_CALL(hipDeviceSynchronize());

  return host_states;
}

[[nodiscard]] State* call_init_global_buffers_kernel(size_t grid_size, size_t block_size,
                                                     uint32_t max_nlefs, uint32_t max_nbarriers) {
  CUDA_CALL(hipDeviceSetLimit(hipLimitMallocHeapSize, 1024ULL * 1024ULL * 2048LL));  // 2GB
  assert(grid_size > 0);                                                               // NOLINT
  // We allocate one state per block

  State* states{nullptr};
  cuda::std::atomic<bool>* dev_status{nullptr};
  cuda::std::atomic<bool> host_status{};

  CUDA_CALL(hipMalloc(&states, sizeof(State) * grid_size));
  CUDA_CALL(hipMalloc(&dev_status, sizeof(cuda::std::atomic<bool>)));
  CUDA_CALL(hipMemset(dev_status, true, sizeof(cuda::std::atomic<bool>)));

  init_global_buffers_kernel<<<grid_size, block_size>>>(states, grid_size, max_nlefs, max_nbarriers,
                                                        dev_status);
  CUDA_CALL(hipDeviceSynchronize());
  CUDA_CALL(hipMemcpy(&host_status, dev_status, sizeof(cuda::std::atomic<bool>),
                       hipMemcpyDeviceToHost));
  CUDA_CALL(hipDeviceSynchronize());
  if (!host_status) {
    throw std::runtime_error("Unable to allocate enough memory to initialize device buffers.");
  }
  return states;
}

void call_free_global_buffers_kernel(size_t grid_size, size_t block_size, State* states) {
  free_global_buffers_kernel<<<grid_size, block_size>>>(states, grid_size);
  CUDA_CALL(hipDeviceSynchronize());
}

}  // namespace modle::cu::Simulation
